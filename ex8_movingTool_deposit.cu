#include "hip/hip_runtime.h"
/**
 * @file ex7_movingSTL.cu
 * @author Dániel NAGY
 * @version 1.0
 * @brief Gravitational deposition example
 * @date 2023.09.12.
 * 
 * This code simulates the deposition of particles with special STL geometry.
 *
*/


#include <iostream>
#include <fstream>
#include <filesystem>
#include <string>
#include <chrono>

constexpr int NumberOfParticles = 131072;
constexpr int NumberOfMaterials = 2;

constexpr int sizeMoving = 0;
constexpr int sizeWalls = 10;
constexpr int NumberOfBoundaries = sizeMoving + sizeWalls;


int NumberOfActiveParticles = 32768;
constexpr int ParticlesPerLayer = 32768;

#include "source/solver.cuh"


int main(int argc, char const *argv[])
{
    //Set GPU
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    //set the initial particle distribution
    struct particleDistribution pdist;
    pdist.min.x = -0.5f;
    pdist.max.x =  0.5f;
    pdist.min.y = -0.35f;
    pdist.max.y =  0.35f;
    pdist.min.z = 0.4f;
    pdist.max.z = 2.2f;
    pdist.vmean = 0.00f;
    pdist.vsigma= 0.00f;
    pdist.Rmean = 4.6e-3f;
    pdist.Rsigma= 0.8e-3f;

    //material parameters
    struct materialParameters materials;
    materials.rho[0]= 1850.0f;
    materials.E[0] = 2.0e6f;
    materials.G[0] = 1.0e6f;
    materials.nu[0] = 0.38f;
    materials.e[0] = 0.5f;
    materials.mu[0] = 0.6f;
    materials.mu0[0] = 0.7f;
    materials.mur[0] = 0.03f;

    //tool parameters
    materials.rho[1]= 4000.0f;
    materials.E[1] = 2.0e8f;
    materials.G[1] = 1.0e8f;
    materials.nu[1] = 0.3f;
    materials.e[1] = 0.2f;
    materials.mu[1] = 0.6f;
    materials.mu0[1] = 0.7f;
    materials.mur[1] = 0.03f;

    materialHandling::calculateMaterialContact(materials,materialHandling::methods::Min,materialHandling::methods::HarmonicMean,materialHandling::methods::HarmonicMean);

    //timestep settings
    float dt = 1.0e-4f;
    float saves = 0.05f;
    struct timestepping timestep(0.0f,20.0f,dt,saves);

    //body forces
    struct bodyForce gravity;
    gravity.x = 0.0f;
    gravity.y = 0.0f;
    gravity.z = -9.81f;

    //BCs - wall on the bottom at z=0 and around in a 2m x 2m domain
    struct boundaryCondition BCsH;
    struct boundaryCondition BCsD;

    ioHandling::readGeometrySTL(BCsH,0,BoundaryConditionType::HertzWall,1,1.0f,"data/ex8_walls.stl");
    ioHandling::readGeometrySTL(BCsH,sizeWalls,BoundaryConditionType::HertzWall,1,1.0f,"data/ex8_tool.stl");

    domainHandling::translateBoundaryConditions(BCsH,sizeWalls,sizeWalls+sizeMoving,-2.6f,0.0f,0.2f,true);
    domainHandling::convertBoundaryConditions(BCsH,BCsD);

    //particles, host side
    struct particle particlesH;
    memoryHandling::allocateHostParticles(particlesH);
    particleHandling::generateParticleLocation(
        particlesH,
        pdist,
        particleHandling::ParticleSizeDistribution::Uniform,
        particleHandling::ParticleVelocityDistribution::Uniform);
    //ioHandling::readParticlesVTK(particlesH,"data/ex8_input_131k.vtu",NumberOfParticles);
    particleHandling::generateParticleParameters(particlesH,materials,0,0,NumberOfParticles);

    //particles, device side
    struct particle particlesD;
    memoryHandling::allocateDeviceParticles(particlesD);
    memoryHandling::synchronizeParticles(particlesD,particlesH,memoryHandling::listOfVariables::All,hipMemcpyHostToDevice);

    //boundary, device and host side of forces
    memoryHandling::allocateDeviceBoundary(BCsH,BCsD);

    //create the output folder
    std::string output_folder = "output";
    if (std::filesystem::exists(output_folder)) 
    {
        std::filesystem::remove_all(output_folder);
    }
    std::filesystem::create_directory(output_folder);

    //create a file to save the energy
    std::ofstream energy(output_folder + "/energy.csv");
    energy << "Kin.\tPot.\tTot.\n";

    //simulation settings
    int numberOfLaunches = (timestep.numberOfSteps+1)/timestep.saveSteps;


    //SIMULATION
    auto startTime = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < numberOfLaunches; i++)
    {
        int GridSize = (NumberOfActiveParticles + 1)/BlockSize;
        //solve
        /*void *kernelArgs[] = {
            (void*)&particlesD,
            (void*)&NumberOfParticles,
            (void*)&materials,
            (void*)&timestep,
            (void*)&gravity,
            (void*)&BCsD,
            (void*)&i
        };
        hipLaunchCooperativeKernel((void*)solver, GridSize, BlockSize, kernelArgs);
        CHECK(hipDeviceSynchronize());*/

        solver<<<GridSize,BlockSize>>>(particlesD,NumberOfActiveParticles,materials,timestep,gravity,BCsD,i);
        CHECK(hipDeviceSynchronize());

        //save
        std::string name = output_folder + "/test_" + std::to_string(i) + ".vtu";
        ioHandling::saveParticlesVTK(NumberOfActiveParticles,particlesH,name);

        //copy D2H
        memoryHandling::synchronizeParticles(
            particlesH,
            particlesD,
            memoryHandling::listOfVariables::Position,
            hipMemcpyDeviceToHost
        );
        memoryHandling::synchronizeParticles(
            particlesH,
            particlesD,
            memoryHandling::listOfVariables::Velocity,
            hipMemcpyDeviceToHost
        );

        //save energy
        float K = forceHandling::calculateTotalKineticEnergy(particlesH,NumberOfActiveParticles);
        float P = forceHandling::calculateTotalPotentialEnergy(particlesH,gravity,NumberOfActiveParticles);
        energy << K << "\t" << P << "\t" << K+P << "\n";

        std::cout << "Launch " << i << "\t/ " << numberOfLaunches << "\n";
        std::cout << "K="<< K << "\t P=" << P << "\t T=" << K+P << "\n";

        if(i > 0 && i%27 == 0)
        {
            NumberOfActiveParticles += ParticlesPerLayer;
            if(NumberOfActiveParticles > NumberOfParticles) NumberOfActiveParticles = NumberOfParticles;

            std::cout << "!!!! Number of particles = " << NumberOfActiveParticles << "\n\n";
        }
        
    }
    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();
    std::cout << "Runtime: " << duration/1000 << " ms" << std::endl;

    energy.flush();
    energy.close();

    memoryHandling::freeHostParticles(particlesH);
    memoryHandling::freeDeviceParticles(particlesD);

}