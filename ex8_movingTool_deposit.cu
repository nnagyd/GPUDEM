#include "hip/hip_runtime.h"
/**
 * @file ex7_movingSTL.cu
 * @author Dániel NAGY
 * @version 1.0
 * @brief Gravitational deposition example
 * @date 2023.09.12.
 * 
 * This code simulates the deposition of particles with special STL geometry.
 *
*/


#include <iostream>
#include <fstream>
#include <filesystem>
#include <string>
#include <chrono>

constexpr int NumberOfParticles = 147456;
constexpr int NumberOfMaterials = 2;

constexpr int sizeWalls = 10;
constexpr int NumberOfBoundaries = sizeWalls;


int NumberOfActiveParticles = 147456;
constexpr int ParticlesPerLayer = 147456;

#include "source/solver.cuh"


int main(int argc, char const *argv[])
{
    //Set GPU
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    //set the initial particle distribution
    struct particleDistribution pdist;
    pdist.min.x = -0.5f;
    pdist.max.x =  0.5f;
    pdist.min.y = -0.35f;
    pdist.max.y =  0.35f;
    pdist.min.z = 0.0f;
    pdist.max.z = 9.0f;
    pdist.vmean = 0.00f;
    pdist.vsigma= 0.00f;
    pdist.Rmean = 4.4e-3f;
    pdist.Rsigma= 0.8e-3f;

    //material parameters
    struct materialParameters materials;
    materials.rho[0]= 1850.0f;
    materials.E[0] = 2.0e6f;
    materials.G[0] = 1.0e6f;
    materials.nu[0] = 0.38f;
    materials.e[0] = 0.5f;
    materials.mu[0] = 0.6f;
    materials.mu0[0] = 0.7f;
    materials.mur[0] = 0.03f;

    //tool parameters
    materials.rho[1]= 4000.0f;
    materials.E[1] = 2.0e8f;
    materials.G[1] = 1.0e8f;
    materials.nu[1] = 0.3f;
    materials.e[1] = 0.2f;
    materials.mu[1] = 0.6f;
    materials.mu0[1] = 0.7f;
    materials.mur[1] = 0.03f;

    materialHandling::calculateMaterialContact(materials,materialHandling::methods::Min,materialHandling::methods::HarmonicMean,materialHandling::methods::HarmonicMean);

    //timestep settings
    float dt = 1.0e-4f;
    float saves = 0.01f;
    struct timestepping timestep(0.0f,20.0f,dt,saves);

    //body forces
    struct bodyForce gravity;
    gravity.x = 0.0f;
    gravity.y = 0.0f;
    gravity.z = -9.81f;

    //BCs - wall on the bottom at z=0 and around in a 2m x 2m domain
    struct boundaryCondition BCsH;
    struct boundaryCondition BCsD;

    ioHandling::readGeometrySTL(BCsH,0,BoundaryConditionType::HertzWall,1,1.0f,"data/ex8_walls.stl");
    domainHandling::convertBoundaryConditions(BCsH,BCsD);

    //particles, host side
    struct particle particlesH;
    memoryHandling::allocateHostParticles(particlesH);
    /*particleHandling::generateParticleLocation(
        particlesH,
        pdist,
        particleHandling::ParticleSizeDistribution::Uniform,
        particleHandling::ParticleVelocityDistribution::Uniform);*/
    ioHandling::readParticlesVTK(particlesH,"data/ex8_input_147k3.vtu",NumberOfParticles);

    std:: cout << "v = " << particlesH.v.x[0] << "\n";
    //sort based on location z
    /*for(int i = 0; i < NumberOfParticles - 1; i++)
    {
        if(i % 25000 == 0) printf("Sorted: %d\n",i);
        for(int j = 0; j < NumberOfActiveParticles - 1 - i; j++)
        {
            if(particlesH.u.z[j] > particlesH.u.z[j+1])
            {
                float x,y,z,R;
                x = particlesH.u.x[j];
                y = particlesH.u.y[j];
                z = particlesH.u.z[j];
                vx = particlesH.u.x[j];
                vy = particlesH.u.y[j];
                vz = particlesH.u.z[j];
                R = particlesH.R[j];

                particlesH.u.x[j] = particlesH.u.x[j+1];
                particlesH.u.y[j] = particlesH.u.y[j+1];
                particlesH.u.z[j] = particlesH.u.z[j+1];
                particlesH.v.x[j] = particlesH.v.x[j+1];
                particlesH.v.y[j] = particlesH.v.y[j+1];
                particlesH.v.z[j] = particlesH.v.z[j+1];
                particlesH.R[j] = particlesH.R[j+1];

                particlesH.u.x[j+1] = x;
                particlesH.u.y[j+1] = y;
                particlesH.u.z[j+1] = z;
                particlesH.v.x[j+1] = vx;
                particlesH.v.y[j+1] = vy;
                particlesH.v.z[j+1] = vz;
                particlesH.R[j+1] = R;
            }
        }
    }*/

    particleHandling::generateParticleParameters(particlesH,materials,0,0,NumberOfParticles);

    //particles, device side
    struct particle particlesD;
    memoryHandling::allocateDeviceParticles(particlesD);
    memoryHandling::synchronizeParticles(particlesD,particlesH,memoryHandling::listOfVariables::All,hipMemcpyHostToDevice);

    //boundary, device and host side of forces
    memoryHandling::allocateDeviceBoundary(BCsH,BCsD);

    //create the output folder
    std::string output_folder = "output";
    if (std::filesystem::exists(output_folder)) 
    {
        std::filesystem::remove_all(output_folder);
    }
    std::filesystem::create_directory(output_folder);

    //create a file to save the energy
    std::ofstream energy(output_folder + "/energy.csv");
    energy << "Kin.\tPot.\tTot.\n";

    //simulation settings
    int numberOfLaunches = (timestep.numberOfSteps+1)/timestep.saveSteps;


    //SIMULATION
    auto startTime = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < numberOfLaunches; i++)
    {
        int GridSize = (NumberOfActiveParticles + 1)/BlockSize;
        //solve
        /*void *kernelArgs[] = {
            (void*)&particlesD,
            (void*)&NumberOfParticles,
            (void*)&materials,
            (void*)&timestep,
            (void*)&gravity,
            (void*)&BCsD,
            (void*)&i
        };
        hipLaunchCooperativeKernel((void*)solver, GridSize, BlockSize, kernelArgs);
        CHECK(hipDeviceSynchronize());*/

        solver<<<GridSize,BlockSize>>>(particlesD,NumberOfActiveParticles,materials,timestep,gravity,BCsD,i);
        CHECK(hipDeviceSynchronize());

        //save
        std::string name = output_folder + "/test_" + std::to_string(i) + ".vtu";
        ioHandling::saveParticlesVTK(NumberOfActiveParticles,particlesH,name);

        //copy D2H
        memoryHandling::synchronizeParticles(
            particlesH,
            particlesD,
            memoryHandling::listOfVariables::Position,
            hipMemcpyDeviceToHost
        );
        memoryHandling::synchronizeParticles(
            particlesH,
            particlesD,
            memoryHandling::listOfVariables::Velocity,
            hipMemcpyDeviceToHost
        );

        //save energy
        float K = forceHandling::calculateTotalKineticEnergy(particlesH,NumberOfActiveParticles);
        float P = forceHandling::calculateTotalPotentialEnergy(particlesH,gravity,NumberOfActiveParticles);
        energy << K << "\t" << P << "\t" << K+P << "\n";

        std::cout << "Launch " << i << "\t/ " << numberOfLaunches << "\n";
        std::cout << "K="<< K << "\t P=" << P << "\t T=" << K+P << "\n";

        if(i > 0 && i%12 == 0)
        {
            NumberOfActiveParticles += ParticlesPerLayer;
            if(NumberOfActiveParticles > NumberOfParticles) NumberOfActiveParticles = NumberOfParticles;

            std::cout << "!!!! Number of particles = " << NumberOfActiveParticles << "\n\n";
        }
        
    }
    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();
    std::cout << "Runtime: " << duration/1000 << " ms" << std::endl;

    energy.flush();
    energy.close();

    memoryHandling::freeHostParticles(particlesH);
    memoryHandling::freeDeviceParticles(particlesD);

}