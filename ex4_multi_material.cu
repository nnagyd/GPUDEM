#include "hip/hip_runtime.h"
/**
 * @file ex1_deposition.cu
 * @author Dániel NAGY
 * @version 1.0
 * @brief Gravitational deposition example
 * @date 2023.09.12.
 * 
 * This code simulates the deposition of a denser and a lighter material.
 * The dense material has rho=1000kg/m3 and light material has rho=200kg/m3
 *
 * Domain
 *  - Layout = 2m x 2m
*/


#include <iostream>
#include <fstream>
#include <filesystem>
#include <string>
#include <chrono>

constexpr int NumberOfParticles = 4096;
constexpr int NumberOfMaterials = 3;
constexpr int NumberOfBoundaries = 5;

#include "source/solver.cuh"


int main(int argc, char const *argv[])
{
    //Set GPU
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    //material parameters
    struct materialParameters materials;
    //material 0
    materials.rho[0]= 200.0f;
    materials.E[0] = 10000.0f;
    materials.G[0] = 10000.0f;
    materials.nu[0] = 0.3f;
    materials.e[0] = 0.001f;
    materials.mu[0] = 0.6f;
    materials.mu0[0] = 0.7f;
    materials.mur[2] = 0.02f;

    //material 2
    materials.rho[1]=1000.0f;
    materials.E[1] = 20000.0f;
    materials.G[1] = 20000.0f;
    materials.nu[1] = 0.3f;
    materials.e[1] = 0.002f;
    materials.mu[1] = 0.4f;
    materials.mu0[1] = 0.5f;
    materials.mur[2] = 0.02f;

    //wall
    materials.E[2] = 200000.0f;
    materials.G[2] = 200000.0f;
    materials.nu[2] = 0.3f;
    materials.e[2] = 0.002f;
    materials.mu[2] = 0.7f;
    materials.mu0[2] = 0.8f;
    materials.mur[2] = 0.05f;

    materialHandling::calculateMaterialContact(materials,materialHandling::methods::Min,materialHandling::methods::HarmonicMean,materialHandling::methods::HarmonicMean);
    materialHandling::printMaterialInfo(materials,true);

    //particle distribution
    struct particleDistribution pdist;
    pdist.min.x = -1.0f;
    pdist.max.x =  1.0f;
    pdist.min.y = -1.0f;
    pdist.max.y =  1.0f;
    pdist.min.z = 0.5f;
    pdist.max.z = 5.0f;
    pdist.vmean = 0.0f;
    pdist.vsigma = 0.00f;
    pdist.Rmean = 0.05f;
    pdist.Rsigma = 0.01f;

    //timestep settings
    float dt = 1e-4f;
    float saves = 0.05f;
    struct timestepping timestep(0.0f,15.0f,dt,saves);

    //body forces
    struct bodyForce gravity;
    gravity.x = 0.0f;
    gravity.y = 0.0f;
    gravity.z = -9.81f;

    //BCs - wall on the bottom at z=0 and around in a 2m x 2m domain
    struct boundaryCondition BCs;
    BCs.n[0] = vec3D(0.0f,0.0f,-1.0f); BCs.p[0] = vec3D(0.0f,0.0f,0.0f);  
    BCs.n[1] = vec3D(-1.0f,0.0f,0.0f); BCs.p[1] = vec3D(-1.0f,0.0f,0.0f);
    BCs.n[2] = vec3D( 1.0f,0.0f,0.0f); BCs.p[2] = vec3D( 1.0f,0.0f,0.0f);
    BCs.n[3] = vec3D(0.0f,-1.0f,0.0f); BCs.p[3] = vec3D(0.0f,-1.0f,0.0f);
    BCs.n[4] = vec3D(0.0f, 1.0f,0.0f); BCs.p[4] = vec3D(0.0f, 1.0f,0.0f);
    for(int i = 0; i < NumberOfBoundaries; i++)
    {
        BCs.type[i] = BoundaryConditionType::HertzWall; 
        BCs.material[i] = 2;
    }

    //particles, host side
    struct particle particlesH;
    memoryHandling::allocateHostParticles(particlesH);
    particleHandling::generateParticleLocation(
        particlesH,
        pdist,
        particleHandling::ParticleSizeDistribution::Uniform,
        particleHandling::ParticleVelocityDistribution::Uniform);
    particleHandling::generateParticleParameters(particlesH,materials,0,0,NumberOfParticles/2);
    particleHandling::generateParticleParameters(particlesH,materials,1,NumberOfParticles/2,NumberOfParticles);

    //particles, device side
    struct particle particlesD;
    memoryHandling::allocateDeviceParticles(particlesD);
    memoryHandling::synchronizeParticles(particlesD,particlesH,memoryHandling::listOfVariables::All,hipMemcpyHostToDevice);

    //create the output folder
    std::string output_folder = "output";
    if (std::filesystem::exists(output_folder)) 
    {
        std::filesystem::remove_all(output_folder);
    }
    std::filesystem::create_directory(output_folder);

    //create a file to save the energy
    std::ofstream energy(output_folder + "/energy.csv");
    energy << "Kin.\tPot.\tTot.\n";

    //simulation settings
    int GridSize = (NumberOfParticles + 1)/BlockSize;
    std::cout << "<<<" << GridSize << "," << BlockSize << ">>>\n";
    int numberOfLaunches = (timestep.numberOfSteps+1)/timestep.saveSteps;

    //SIMULATION
    auto startTime = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < numberOfLaunches; i++)
    {        
        //save energy
        float K = forceHandling::calculateTotalKineticEnergy(particlesH,NumberOfParticles);
        float P = forceHandling::calculateTotalPotentialEnergy(particlesH,gravity,NumberOfParticles);
        energy << K << "\t" << P << "\t" << K+P << "\n";


        //print info
        if(i%10==0)
        {
            std::cout << "Launch " << i << "\t/ " << numberOfLaunches << "\n";
            std::cout << "K="<< K << "\t P=" << P << "\t T=" << K+P << "\n";
        }

        if(i == 100)
        {
            gravity.x = 3.0f;
            gravity.z = 12.0f;
        }
        if(i == 105)
        {
            gravity.x = -3.0f;
            gravity.z = -12.0f;
        }
        if(i == 110)
        {
            gravity.x = 0.0f;
            gravity.z = -9.81f;
        }
        if(i == 150)
        {
            gravity.x = 3.0f;
            gravity.z = 12.0f;
        }
        if(i == 155)
        {
            gravity.x = -3.0f;
            gravity.z = -12.0f;
        }
        if(i == 160)
        {
            gravity.x = 0.0f;
            gravity.z = -9.81f;
        }

        //save
        std::string name = output_folder + "/test_" + std::to_string(i) + ".vtu";
        ioHandling::saveParticlesVTK(NumberOfParticles,particlesH,name);

        //solve
        void *kernelArgs[] = {
            (void*)&particlesD,
            (void*)&NumberOfParticles,
            (void*)&materials,
            (void*)&timestep,
            (void*)&gravity,
            (void*)&BCs,
            (void*)&i
        };
        hipLaunchCooperativeKernel((void*)solver, GridSize, BlockSize, kernelArgs);
        CHECK(hipDeviceSynchronize());

        //copy D2H
        memoryHandling::synchronizeParticles(
            particlesH,
            particlesD,
            memoryHandling::listOfVariables::Position,
            hipMemcpyDeviceToHost
        );
        memoryHandling::synchronizeParticles(
            particlesH,
            particlesD,
            memoryHandling::listOfVariables::Velocity,
            hipMemcpyDeviceToHost
        );
    }
    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();
    std::cout << "Runtime: " << duration/1000 << " ms" << std::endl;

    energy.flush();
    energy.close();

    memoryHandling::freeHostParticles(particlesH);
    memoryHandling::freeDeviceParticles(particlesD);

}