/**
 * @file ex7_movingSTL.cu
 * @author Dániel NAGY
 * @version 1.0
 * @brief Gravitational deposition example
 * @date 2023.09.12.
 * 
 * This code simulates the deposition of particles with special STL geometry.
 *
*/


#include <iostream>
#include <fstream>
#include <filesystem>
#include <string>
#include <chrono>

constexpr int NumberOfParticles = 16384;
constexpr int NumberOfMaterials = 3;

constexpr int sizeMoving = 10;
constexpr int sizeWalls = 10;
constexpr int NumberOfBoundaries = sizeMoving + sizeWalls;

#include "source/solver.cuh"


int main(int argc, char const *argv[])
{
    //Set GPU
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    //material parameters
    struct materialParameters materials;
    materials.rho[0]= 300.0f;
    materials.E[0] = 10000.0f;
    materials.G[0] = 10000.0f;
    materials.nu[0] = 0.3f;
    materials.e[0] = 0.001f;
    materials.mu[0] = 0.6f;
    materials.mu0[0] = 0.7f;
    materials.mur[0] = 0.03f;

    //walls
    materials.rho[1]= 1000.0f;
    materials.E[1] = 200000.0f;
    materials.G[1] = 200000.0f;
    materials.nu[1] = 0.3f;
    materials.e[1] = 0.1f;
    materials.mu[1] = 0.6f;
    materials.mu0[1] = 0.7f;
    materials.mur[1] = 0.03f;

    //stl
    materials.rho[2]= 4000.0f;
    materials.E[2] = 2000000.0f;
    materials.G[2] = 2000000.0f;
    materials.nu[2] = 0.3f;
    materials.e[2] = 0.2f;
    materials.mu[2] = 0.6f;
    materials.mu0[2] = 0.7f;
    materials.mur[2] = 0.03f;

    materialHandling::calculateMaterialContact(materials,materialHandling::methods::Min,materialHandling::methods::HarmonicMean,materialHandling::methods::HarmonicMean);
    materialHandling::printMaterialInfo(materials,true);

    //timestep settings
    float dt = 5e-5f;
    float saves = 0.0005f;
    struct timestepping timestep(0.0f,10.0f,dt,saves);

    //body forces
    struct bodyForce gravity;
    gravity.x = 0.0f;
    gravity.y = 0.0f;
    gravity.z = -9.81f;

    //BCsH - wall on the bottom at z=0 and around in a 2m x 2m domain
    struct boundaryCondition BCsH;
    struct boundaryCondition BCsD;

    ioHandling::readGeometrySTL(BCsH,0,BoundaryConditionType::HertzWall,1,1.0f,"data/ex7_walls.stl");
    ioHandling::readGeometrySTL(BCsH,sizeWalls,BoundaryConditionType::HertzWall,2,1.0f,"data/ex7_moving.stl");

    domainHandling::printBoundaryConditions(BCsH);
    domainHandling::convertBoundaryConditions(BCsH,BCsD);
    domainHandling::translateBoundaryConditions(BCsH,sizeWalls,sizeWalls+sizeMoving,0.0f,0.0f,0.0f);

    //particles, host side
    struct particle particlesH;
    memoryHandling::allocateHostParticles(particlesH);
    ioHandling::readParticlesVTK(particlesH,"data/ex7_input.vtu");
    particleHandling::generateParticleParameters(particlesH,materials,0,0,NumberOfParticles);

    //particles, device side
    struct particle particlesD;
    memoryHandling::allocateDeviceParticles(particlesD);
    memoryHandling::synchronizeParticles(particlesD,particlesH,memoryHandling::listOfVariables::All,hipMemcpyHostToDevice);

    //boundary, device and host side of forces
    memoryHandling::allocateDeviceBoundary(BCsD);
    BCsH.F = new vec3D[NumberOfBoundaries];

    //create the output folder
    std::string output_folder = "output";
    if (std::filesystem::exists(output_folder)) 
    {
        std::filesystem::remove_all(output_folder);
    }
    std::filesystem::create_directory(output_folder);

    //create a file to save the energy
    std::ofstream energy(output_folder + "/energy.csv");
    energy << "Kin.\tPot.\tTot.\n";

    //simulation settings
    int GridSize = (NumberOfParticles + 1)/BlockSize;
    std::cout << "<<<" << GridSize << "," << BlockSize << ">>>\n";
    int numberOfLaunches = (timestep.numberOfSteps+1)/timestep.saveSteps;

    //SIMULATION
    auto startTime = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < numberOfLaunches; i++)
    { 
        //solve
        void *kernelArgs[] = {
            (void*)&particlesD,
            (void*)&NumberOfParticles,
            (void*)&materials,
            (void*)&timestep,
            (void*)&gravity,
            (void*)&BCsH,
            (void*)&i
        };
        hipLaunchCooperativeKernel((void*)solver, GridSize, BlockSize, kernelArgs);
        CHECK(hipDeviceSynchronize());

        if(i % 50 == 0)
        {      
            //save
            std::string name = output_folder + "/test_" + std::to_string(i) + ".vtu";
            std::string name2 = output_folder + "/test_" + std::to_string(i) + ".stl";
            ioHandling::saveParticlesVTK(NumberOfParticles,particlesH,name);
            ioHandling::writeGeometrySTL(BCsH,sizeWalls,sizeWalls+sizeMoving,name2);

            //copy D2H
            memoryHandling::synchronizeParticles(
                particlesH,
                particlesD,
                memoryHandling::listOfVariables::Position,
                hipMemcpyDeviceToHost
            );
            memoryHandling::synchronizeParticles(
                particlesH,
                particlesD,
                memoryHandling::listOfVariables::Velocity,
                hipMemcpyDeviceToHost
            );
            memoryHandling::synchronizeBoundary(BCsH.F,BCsD);

            //save energy
            float K = forceHandling::calculateTotalKineticEnergy(particlesH,NumberOfParticles);
            float P = forceHandling::calculateTotalPotentialEnergy(particlesH,gravity,NumberOfParticles);
            energy << K << "\t" << P << "\t" << K+P << "\n";

            std::cout << "Launch " << i << "\t/ " << numberOfLaunches << "\n";
            std::cout << "K="<< K << "\t P=" << P << "\t T=" << K+P << "\n";

            //save force
            vec3D Fsum = vec3D(0.0f,0.0f,0.0f);
            for(int j = sizeMoving; j < NumberOfBoundaries; j++)
            {
                Fsum = Fsum + BCsH.F[j];
                //std::cout << "  -> F[" << j <<"] = (" << F[j].x << "," << F[j].y << "," << F[j].z << ")\n";
            }
            std::cout << "F = (" << Fsum.x << "," << Fsum.y << "," << Fsum.z << ")\n";
        }
        
        if(true)
        {
            domainHandling::translateBoundaryConditions(BCsH,sizeWalls,sizeWalls+sizeMoving,-0.0005f,0.0f,0.0f);
            domainHandling::translateBoundaryConditions(BCsD,sizeWalls,sizeWalls+sizeMoving,-0.0005f,0.0f,0.0f);
        }
        
    }
    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();
    std::cout << "Runtime: " << duration/1000 << " ms" << std::endl;

    energy.flush();
    energy.close();

    memoryHandling::freeHostParticles(particlesH);
    memoryHandling::freeDeviceParticles(particlesD);

}