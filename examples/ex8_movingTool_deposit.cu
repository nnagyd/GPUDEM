#include "hip/hip_runtime.h"
/**
 * @file ex7_movingSTL.cu
 * @author Dániel NAGY
 * @version 1.0
 * @brief Gravitational deposition example
 * @date 2023.09.12.
 * 
 * This code simulates the deposition of particles with special STL geometry.
 *
*/


#include <iostream>
#include <fstream>
#include <filesystem>
#include <string>
#include <chrono>

constexpr int NumberOfParticles = 163840;
constexpr int NumberOfMaterials = 2;

constexpr int sizeWalls = 10;
constexpr int NumberOfBoundaries = sizeWalls;


int NumberOfActiveParticles = 163840;
constexpr int ParticlesPerLayer = 163840;

#include "source/solver.cuh"


int main(int argc, char const *argv[])
{
    //Set GPU
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    //set the initial particle distribution
    struct particleDistribution pdist;
    pdist.min.x = -0.5f;
    pdist.max.x =  0.5f;
    pdist.min.y = -0.3f;
    pdist.max.y =  0.3f;
    pdist.min.z = 0.0f;
    pdist.max.z = 1.9f;
    pdist.vmean = 0.00f;
    pdist.vsigma= 0.00f;
    pdist.Rmean = 5.0e-3f;
    pdist.Rsigma= 1.0e-3f;

    //material parameters
    struct materialParameters materials;
    materials.rho[0]= 1850.0f;
    materials.E[0] = 2.0e6f;
    materials.G[0] = 1.0e6f;
    materials.nu[0] = 0.38f;
    materials.e[0] = 0.5f;
    materials.mu[0] = 0.6f;
    materials.mu0[0] = 0.7f;
    materials.mur[0] = 0.03f;

    //tool parameters
    materials.rho[1]= 4000.0f;
    materials.E[1] = 2.0e8f;
    materials.G[1] = 1.0e8f;
    materials.nu[1] = 0.3f;
    materials.e[1] = 0.2f;
    materials.mu[1] = 0.6f;
    materials.mu0[1] = 0.7f;
    materials.mur[1] = 0.03f;

    materialHandling::calculateMaterialContact(materials,materialHandling::methods::Min,materialHandling::methods::HarmonicMean,materialHandling::methods::HarmonicMean);

    //timestep settings
    float dt = 1e-4f;
    float saves = 0.1f;
    struct timestepping timestep(0.0f,20.1f,dt,saves);

    //body forces
    struct bodyForce gravity;
    gravity.x = 0.0f;
    gravity.y = 0.0f;
    gravity.z = -9.81f;

    //BCs - wall on the bottom at z=0 and around in a 2m x 2m domain
    struct boundaryCondition BCsH;
    struct boundaryCondition BCsD;

    ioHandling::readGeometrySTL(BCsH,0,BoundaryConditionType::HertzWall,1,1.0f,"data/ex8_walls.stl");
    domainHandling::convertBoundaryConditions(BCsH,BCsD);

    //particles, host side
    struct particle particlesH;
    memoryHandling::allocateHostParticles(particlesH);
    particleHandling::generateParticleLocation(
        particlesH,
        pdist,
        particleHandling::ParticleSizeDistribution::Uniform,
        particleHandling::ParticleVelocityDistribution::Uniform);
    //ioHandling::readParticlesVTK(particlesH,"data/ex8_input_147k3.vtu",NumberOfParticles);

    std:: cout << "v = " << particlesH.v.x[0] << "\n";
    //sort based on location z
    /*for(int i = 0; i < NumberOfParticles - 1; i++)
    {
        if(i % 25000 == 0) printf("Sorted: %d\n",i);
        for(int j = 0; j < NumberOfActiveParticles - 1 - i; j++)
        {
            if(particlesH.u.z[j] > particlesH.u.z[j+1])
            {
                float x,y,z,R;
                x = particlesH.u.x[j];
                y = particlesH.u.y[j];
                z = particlesH.u.z[j];
                vx = particlesH.u.x[j];
                vy = particlesH.u.y[j];
                vz = particlesH.u.z[j];
                R = particlesH.R[j];

                particlesH.u.x[j] = particlesH.u.x[j+1];
                particlesH.u.y[j] = particlesH.u.y[j+1];
                particlesH.u.z[j] = particlesH.u.z[j+1];
                particlesH.v.x[j] = particlesH.v.x[j+1];
                particlesH.v.y[j] = particlesH.v.y[j+1];
                particlesH.v.z[j] = particlesH.v.z[j+1];
                particlesH.R[j] = particlesH.R[j+1];

                particlesH.u.x[j+1] = x;
                particlesH.u.y[j+1] = y;
                particlesH.u.z[j+1] = z;
                particlesH.v.x[j+1] = vx;
                particlesH.v.y[j+1] = vy;
                particlesH.v.z[j+1] = vz;
                particlesH.R[j+1] = R;
            }
        }
    }*/

    particleHandling::generateParticleParameters(particlesH,materials,0,0,NumberOfParticles);

    //particles, device side
    struct particle particlesD;
    memoryHandling::allocateDeviceParticles(particlesD);
    memoryHandling::synchronizeParticles(particlesD,particlesH,memoryHandling::listOfVariables::All,hipMemcpyHostToDevice);

    //boundary, device and host side of forces
    memoryHandling::allocateDeviceBoundary(BCsH,BCsD);

    //create the output folder
    std::string output_folder = "output";
    if (std::filesystem::exists(output_folder)) 
    {
        std::filesystem::remove_all(output_folder);
    }
    std::filesystem::create_directory(output_folder);

    //create a file to save the energy
    std::ofstream energy(output_folder + "/energy.csv");
    energy << "Kin.\tPot.\tTot.\n";

    //simulation settings
    int numberOfLaunches = (timestep.numberOfSteps+1)/timestep.saveSteps;


    //SIMULATION
    auto startTime = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < numberOfLaunches; i++)
    {
        //std::cout << "i = " << i << "\n";
        int GridSize = (NumberOfActiveParticles + 1)/BlockSize;
        //solve
        solver<<<GridSize,BlockSize>>>(particlesD,NumberOfActiveParticles,materials,timestep,gravity,BCsD,i);
        CHECK(hipDeviceSynchronize());

        if(i % 1 == 0)
        {
            std::string name = output_folder + "/test_" + std::to_string(i) + ".vtu";
            ioHandling::saveParticlesVTK(NumberOfActiveParticles,particlesH,name);
    
            //copy D2H
            memoryHandling::synchronizeParticles(
                particlesH,
                particlesD,
                memoryHandling::listOfVariables::Position,
                hipMemcpyDeviceToHost
            );
            memoryHandling::synchronizeParticles(
                particlesH,
                particlesD,
                memoryHandling::listOfVariables::Velocity,
                hipMemcpyDeviceToHost
            );
            memoryHandling::synchronizeParticles(
                particlesH,
                particlesD,
                memoryHandling::listOfVariables::AngularVelocity,
                hipMemcpyDeviceToHost
            );

            //save energy
            float K = forceHandling::calculateTotalKineticEnergy(particlesH,NumberOfActiveParticles);
            float P = forceHandling::calculateTotalPotentialEnergy(particlesH,gravity,NumberOfActiveParticles);
            energy << K << "\t" << P << "\t" << K+P << "\n";

            std::cout << "Launch " << i << "\t/ " << numberOfLaunches << "\n";
            std::cout << "K="<< K << "\t P=" << P << "\t T=" << K+P << "\n";
        }

        if(i % 75000 == 0 && i != 0)
        {
            NumberOfActiveParticles += ParticlesPerLayer;
            if(NumberOfActiveParticles > NumberOfParticles)
            {
                NumberOfActiveParticles = NumberOfParticles;
            }
        }
    }
    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();
    std::cout << "Runtime: " << duration/1000 << " ms" << std::endl;

    energy.flush();
    energy.close();

    memoryHandling::freeHostParticles(particlesH);
    memoryHandling::freeDeviceParticles(particlesD);

}